#include "hip/hip_runtime.h"
/*
count the number of tuple matching criteria for join

*/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tuple.h"


extern "C" {

__global__
void count(
          TUPLE *lt,
          TUPLE *rt,
          int *count,
          int ltn,
          int rtn
          ) 

{

  int j;
    
  //i,jの方向を間違えないように
  /*
   *x軸が縦の方向、y軸が横の方向だよ。
   *だから、xがleft、yがrightに対応しているよ
   */

  //i:the number of y element k:the number of one brefore x element * the total y element
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.x * gridDim.y * blockDim.y;

  
  /*
    transport tuple data to shared memory from global memory
   */

  __shared__ TUPLE Tleft[BLOCK_SIZE_X];
  if(threadIdx.y==0){
    for(j=0;(j<BLOCK_SIZE_X)&&((j+BLOCK_SIZE_X*blockIdx.x)<ltn);j++){
      Tleft[j] = lt[j + BLOCK_SIZE_X * blockIdx.x];
    }
  }

  __syncthreads();  

  TUPLE Tright = rt[i];

  /*
    count loop
   */
  int ltn_g = ltn;
  int rtn_g = rtn;
  uint mcount = 0;

  if(i<rtn_g){
    for(j = 0; j<BLOCK_SIZE_X &&((j+BLOCK_SIZE_X*blockIdx.x)<ltn_g);j++){
      if((Tleft[j].val==Tright.val)) {
        mcount++;
      }
    }
  }    
  count[i + k] = mcount;

}

}
