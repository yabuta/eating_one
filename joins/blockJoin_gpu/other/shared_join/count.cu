#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tuple.h"


extern "C" {

__global__
void count(
          TUPLE *lt,
          TUPLE *rt,
          int *count,
          int ltn,
          int rtn
          ) 

{
  //i,jの方向を間違えないように
  /*
   *x軸が縦の方向、y軸が横の方向だよ。
   *だから、xがleft、yがrightに対応しているよ
   */

  
  int j ;//= blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  for(j = blockIdx.x * BLOCK_SIZE_X; j<(blockIdx.x+1) * BLOCK_SIZE_X && (j<ltn);j++){

    if(i<rtn){

      if((lt[j].val[0]==rt[i].val[0])) {
      
      int n = j * (rtn) + i;
      
        //条件に合致する場合、countを+1する。
        //if corresponding , count += 1 
        count[n] = 1;
      }
    }
      
  }
}

}
