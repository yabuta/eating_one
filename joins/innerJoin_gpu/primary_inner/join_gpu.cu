#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tuple.h"

extern "C" {
__global__ void join(
          TUPLE *lt,
          TUPLE *rt,
          RESULT *jt,
          int *count,
          BUCKET *bucket,
          int *buck_array,
          int *idxcount,
          int left,
          int right
          ) 
{


  int y = blockIdx.y * blockDim.y + threadIdx.y;


  int writeloc = 0;
  if(y!=0){
    writeloc = count[y-1];
  }

  if(y < right){
    int idx = rt[y].val % NB_BKT_ENT;
    if(buck_array[idx] != -1){
      int i = 0;
      for(int k = 0; k < idxcount[idx]; k++){
        if(bucket[buck_array[idx] + k].val == rt[y].val){
          jt[writeloc + i].rkey = rt[y].key;
          jt[writeloc + i].rval = rt[y].val;
          jt[writeloc + i].lkey = lt[bucket[buck_array[idx] + k].adr].key;
          jt[writeloc + i].lval = lt[bucket[buck_array[idx] + k].adr].val;
          
          i++;
          //printf("%d %d\n",jt[count[i] + k].rkey,jt[count[i] + k].lkey);
        }
      }
    }
  }



  //shared memory experience
  /*
  __shared__ int ba[NB_BKT_ENT];

  for(int i=0; i<NB_BKT_ENT ;i++){
    ba[i] = buck_array[i];
  }

  __syncthreads();

  int writeloc = 0;
  if(y!=0){
    writeloc = count[y-1];
  }

  if(y < right){
    int idx = rt[y].val % NB_BKT_ENT;
    if(ba[idx] != -1){
      int i = 0;
      for(int k = 0; k < idxcount[idx]; k++){
        if(bucket[ba[idx] + k].val == rt[y].val){
          jt[writeloc + i].rkey = rt[y].key;
          jt[writeloc + i].rval = rt[y].val;
          jt[writeloc + i].lkey = lt[bucket[ba[idx] + k].adr].key;
          jt[writeloc + i].lval = lt[bucket[ba[idx] + k].adr].val;
          
          i++;
          //printf("%d %d\n",jt[count[i] + k].rkey,jt[count[i] + k].lkey);
        }
      }
    }
  }

  */
    
}    

}
