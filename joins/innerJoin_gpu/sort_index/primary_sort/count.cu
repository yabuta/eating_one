#include "hip/hip_runtime.h"
/*
count the number of tuple matching criteria for join

*/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "tuple.h"

extern "C" {

__device__
uint search(BUCKET *b,int num,uint right){
  uint m,l,r;
  l=0;
  r=right-1;
  do{
    m=(l+r)/2;
    if(num < b[m].val)r=m-1;else l=m+1;
  }while(l<=r&&num!=b[m].val);

  return m;
}


__global__
void count(
          TUPLE *lt,
          uint *count,
          BUCKET *bucket,
          int right,
          int left
          ) 

{
    
  //i,jの方向を間違えないように
  /*
   *x軸が縦の方向、y軸が横の方向だよ。
   *だから、xがleft、yがrightに対応しているよ
   */

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(x < left){
    int idx = lt[x].val;
    uint temp = 0;
    uint bidx = search(bucket,idx,right);
    uint seq = bidx;
    while(bucket[seq].val == idx){
      temp++;
      if(seq == 0) break;
      seq--;
    }
    seq = bidx+1;
    while(bucket[seq].val == idx){
      temp++;
      if(seq == right-1) break;
      seq++;
    }
    count[x] = temp;
  }

}

}
